#define _CRT_SECURE_NO_WARNINGS
#define _CRT_SECURE_NO_DEPRECATE
// include the layer, this is where the actual implementations are

#include "updater_impl-inl.hpp"
// specialize the gpu implementation here
namespace textnet {
namespace updater {
template<>
Updater<gpu, 1>* CreateUpdater<gpu, 1>(UpdaterType type, std::map<std::string, SettingV> &setting, 
                      mshadow::Random<gpu>* prnd) {
  return CreateUpdater_<gpu, 1>(type, setting, prnd); 
}
template<>
Updater<gpu, 2>* CreateUpdater<gpu, 2>(UpdaterType type, std::map<std::string, SettingV> &setting, 
                      mshadow::Random<gpu>* prnd) {
  return CreateUpdater_<gpu, 2>(type, setting, prnd); 
}
template<>
Updater<gpu, 3>* CreateUpdater<gpu, 3>(UpdaterType type, std::map<std::string, SettingV> &setting, 
                      mshadow::Random<gpu>* prnd) {
  return CreateUpdater_<gpu, 3>(type, setting, prnd); 
}
template<>
Updater<gpu, 4>* CreateUpdater<gpu, 4>(UpdaterType type, std::map<std::string, SettingV> &setting, 
                      mshadow::Random<gpu>* prnd) {
  return CreateUpdater_<gpu, 4>(type, setting, prnd); 
}
}  // namespace updater
}  // namespace textnet