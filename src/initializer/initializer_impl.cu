#define _CRT_SECURE_NO_WARNINGS
#define _CRT_SECURE_NO_DEPRECATE
// include the layer, this is where the actual implementations are

#include "initializer_impl-inl.hpp"
// specialize the cpu implementation here
namespace textnet {
namespace initializer {
template<>
Initializer<gpu, 1>* CreateInitializer<gpu, 1>(
                             InitType type,
                             std::map<std::string, SettingV> &setting,
                             mshadow::Random<xpu>* prnd) {
  return CreateInitializer_<gpu, 1>(type, setting, prnd); 
}
template<>
Initializer<gpu, 2>* CreateInitializer<gpu, 2>(
                             InitType type,
                             std::map<std::string, SettingV> &setting,
                             mshadow::Random<xpu>* prnd) {
  return CreateInitializer_<gpu, 2>(type, setting, prnd); 
}
template<>
Initializer<gpu, 3>* CreateInitializer<gpu, 3>(
                             InitType type,
                             std::map<std::string, SettingV> &setting,
                             mshadow::Random<xpu>* prnd) {
  return CreateInitializer_<gpu, 3>(type, setting, prnd); 
}
template<>
Initializer<gpu, 4>* CreateInitializer<gpu, 4>(
                             InitType type,
                             std::map<std::string, SettingV> &setting,
                             mshadow::Random<xpu>* prnd) {
  return CreateInitializer_<gpu, 4>(type, setting, prnd); 
}
}  // namespace initializer
}  // namespace textnet
