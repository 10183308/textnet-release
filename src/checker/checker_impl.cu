#define _CRT_SECURE_NO_WARNINGS
#define _CRT_SECURE_NO_DEPRECATE
// include the layer, this is where the actual implementations are

#include "checker_impl-inl.hpp"
// specialize the gpu implementation here
namespace textnet {
namespace checker {
template<>
Checker<gpu>* CreateChecker<gpu>() {
  return CreateChecker_<gpu>(); 
}

}  // namespace checker
}  // namespace textnet
